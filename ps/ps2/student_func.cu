#include "hip/hip_runtime.h"
// That is instead of RGBARGBARGBARGBA... we suggest transforming to three
// arrays (as in the previous homework we ignore the alpha channel again):
//  1) RRRRRRRR...
//  2) GGGGGGGG...
//  3) BBBBBBBB...
//
// The original layout is known an Array of Structures (AoS) whereas the
// format we are converting to is known as a Structure of Arrays (SoA).

// As a warm-up, we will ask you to write the kernel that performs this
// separation. You should then write the "meat" of the assignment,
// which is the kernel that performs the actual blur. We provide code that
// re-combines your blurred results for each color channel.

//****************************************************************************

// You must fill in the gaussian_blur kernel to perform the blurring of the
// inputChannel, using the array of weights, and put the result in the outputChannel.

// Here is an example of computing a blur, using a weighted average, for a single
// pixel in a small image.
//
// Array of weights:
//
//  0.0  0.2  0.0
//  0.2  0.2  0.2
//  0.0  0.2  0.0
//
// Image (note that we align the array of weights to the center of the box):
//
//    1  2  5  2  0  3
//       -------
//    3 |2  5  1| 6  0       0.0*2 + 0.2*5 + 0.0*1 +
//      |       |
//    4 |3  6  2| 1  4   ->  0.2*3 + 0.2*6 + 0.2*2 +   ->  3.2
//      |       |
//    0 |4  0  3| 4  2       0.0*4 + 0.2*0 + 0.0*3
//       -------
//    9  6  5  0  3  9
//
//         (1)                         (2)                 (3)
//
// A good starting place is to map each thread to a pixel as you have before.
// Then every thread can perform steps 2 and 3 in the diagram above
// completely independently of one another.

// Note that the array of weights is square, so its height is the same as its width.
// We refer to the array of weights as a filter, and we refer to its width with the
// variable filterWidth.

//****************************************************************************

// Your homework submission will be evaluated based on correctness and speed.
// We test each pixel against a reference solution. If any pixel differs by
// more than some small threshold value, the system will tell you that your
// solution is incorrect, and it will let you try again.

// Once you have gotten that working correctly, then you can think about using
// shared memory and having the threads cooperate to achieve better performance.

//****************************************************************************

// Also note that we've supplied a helpful debugging function called checkCudaErrors.
// You should wrap your allocation and copying statements like we've done in the
// code we're supplying you. Here is an example of the unsafe way to allocate
// memory on the GPU:
//
// hipMalloc(&d_red, sizeof(unsigned char) * numRows * numCols);
//
// Here is an example of the safe way to do the same thing:
//
// checkCudaErrors(hipMalloc(&d_red, sizeof(unsigned char) * numRows * numCols));
//
// Writing code the safe way requires slightly more typing, but is very helpful for
// catching mistakes. If you write code the unsafe way and you make a mistake, then
// any subsequent kernels won't compute anything, and it will be hard to figure out
// why. Writing code the safe way will inform you as soon as you make a mistake.

// Finally, remember to free the memory you allocate at the end of the function.

//****************************************************************************

#include "utils.h"
#include <stdio.h>


__global__
void gaussian_blur(const unsigned char* const inputChannel,
                   unsigned char* const outputChannel,
                   int numRows, int numCols,
                   const float* const filter, const int filterWidth)
{
  float res = 0.0;

  int px = threadIdx.x + blockIdx.x*blockDim.x;
  int py = threadIdx.y + blockIdx.y*blockDim.y;
  int ps = py*numCols+px;

//  if (blockIdx.x != 200 || blockIdx.y != 200)
//    return;

  if (px >= numCols || py >=  numRows)
    return;

  for (int i = 0; i < filterWidth; i++)
  {
    for(int j = 0; j < filterWidth; j++)
    {
      int fx = px - filterWidth/2 + i;
      int fy = py - filterWidth/2 + j;
      fx = (0 <= fx)      ? fx : 0;
      fx = (fx < numCols) ? fx : numCols;
      fy = (0 < fy)       ? fy : 0;
      fy = (fy < numRows) ? fy : numRows;
      int pc = fx+fy*numCols;
      int pf = j*filterWidth+i;
      res += inputChannel[pc]*filter[pf];
    }
  }
  outputChannel[ps] = (int)res % 256;
}


__global__
void separateChannels(const uchar4* const inputImageRGBA,
                      int numRows,
                      int numCols,
                      unsigned char* const redChannel,
                      unsigned char* const greenChannel,
                      unsigned char* const blueChannel)
{
  int px = threadIdx.x + blockIdx.x*blockDim.x;
  int py = threadIdx.y + blockIdx.y*blockDim.y;
  int ps = py*numCols+px;

  if (px >= numCols || py >=  numRows)
    return;

  redChannel[ps]   = inputImageRGBA[ps].x;
  greenChannel[ps] = inputImageRGBA[ps].y;
  blueChannel[ps]  = inputImageRGBA[ps].z;
}

__global__
void recombineChannels(const unsigned char* const redChannel,
                       const unsigned char* const greenChannel,
                       const unsigned char* const blueChannel,
                       uchar4* const outputImageRGBA,
                       int numRows,
                       int numCols)
{
  const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                        blockIdx.y * blockDim.y + threadIdx.y);

  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

  if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
    return;

  unsigned char red   = redChannel[thread_1D_pos];
  unsigned char green = greenChannel[thread_1D_pos];
  unsigned char blue  = blueChannel[thread_1D_pos];

  //Alpha should be 255 for no transparency
  uchar4 outputPixel = make_uchar4(red, green, blue, 255);

  outputImageRGBA[thread_1D_pos] = outputPixel;
}

unsigned char *d_red, *d_green, *d_blue;
float         *d_filter;

void allocateMemoryAndCopyToGPU(const size_t numRowsImage, const size_t numColsImage,
                                const float* const h_filter, const size_t filterWidth)
{
  checkCudaErrors(hipMalloc(&d_red,   sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_green, sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_blue,  sizeof(unsigned char) * numRowsImage * numColsImage));

  int size = filterWidth*filterWidth*sizeof(float);
  checkCudaErrors(hipMalloc(&d_filter, size));
  checkCudaErrors(hipMemcpy(d_filter, h_filter, size, hipMemcpyHostToDevice));
}

void your_gaussian_blur(const uchar4 * const h_inputImageRGBA, uchar4 * const d_inputImageRGBA,
                        uchar4* const d_outputImageRGBA, const size_t numRows, const size_t numCols,
                        unsigned char *d_redBlurred, 
                        unsigned char *d_greenBlurred, 
                        unsigned char *d_blueBlurred,
                        const int filterWidth)
{
  const dim3 blockSize(16, 16, 1);
  const dim3 gridSize(numCols/blockSize.x+1, numRows/blockSize.y+1, 1);
//  const dim3 blockSize(1, 1, 1);
//  const dim3 gridSize(1, 1, 1);


  //TODO: Launch a kernel for separating the RGBA image into different color channels

  separateChannels<<<gridSize, blockSize>>>(d_inputImageRGBA, numRows, numCols, d_red, d_green, d_blue);
  checkCudaErrors(hipDeviceSynchronize()); checkCudaErrors(hipGetLastError());

  gaussian_blur<<<gridSize, blockSize>>>(d_red,   d_redBlurred,   numRows, numCols, d_filter, filterWidth);
  gaussian_blur<<<gridSize, blockSize>>>(d_green, d_greenBlurred, numRows, numCols, d_filter, filterWidth);
  gaussian_blur<<<gridSize, blockSize>>>(d_blue,  d_blueBlurred,  numRows, numCols, d_filter, filterWidth);
  checkCudaErrors(hipDeviceSynchronize()); checkCudaErrors(hipGetLastError());

  recombineChannels<<<gridSize, blockSize>>>(d_redBlurred,
                                             d_greenBlurred,
                                             d_blueBlurred,
                                             d_outputImageRGBA,
                                             numRows,
                                             numCols);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

}


//Free all the memory that we allocated
//TODO: make sure you free any arrays that you allocated
void cleanup() {
  checkCudaErrors(hipFree(d_red));
  checkCudaErrors(hipFree(d_green));
  checkCudaErrors(hipFree(d_blue));

  checkCudaErrors(hipFree(d_filter));
}
